#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cstdlib>
#include <time.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// declaration, forward
void pi(int argc, char **argv);


// some hints here
// http://stackoverflow.com/questions/11832202/cuda-random-number-generating
//
// this is pretty good...look at multi core implementation at bottom
// http://cs.umw.edu/~finlayson/class/fall16/cpsc425/notes/cuda-random.html
//
// good article
// http://stackoverflow.com/questions/26650391/generate-random-number-within-a-function-with-hiprand-without-preallocation



/********************** DEVICE CODE *********************/

__global__ void init_random(unsigned int seed, hiprandState_t *states)  {
    hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

__global__ void randoms(hiprandState_t *states, double *numbers)  {
    numbers[blockIdx.x] = hiprand_uniform_double(&states[blockIdx.x]);
}

__global__ void random(double *result)  {

    hiprandState_t state;  // need a local state for each thread, see cs.umw.edu

    hiprand_init(1234ULL, threadIdx.x, 0, &state);
// arg1 -> the seed controls the sequence of random values that are produced
// arg2 -> sequence number is only important with multiple cores
// arg3 -> offset, how much extra we advance in the sequence for each call, can be 0 
// hiprand works like rand - except that it takes a state as a parameter 
    
    double tmp = hiprand_uniform_double(&state);
    result[threadIdx.x] = tmp;
}

/*
__global__ void threadBlockAdd(int *a, int *b, int *c, int n)  {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index];
}
*/

/********************** HOST CODE *********************/

void pi(int argc, char **argv)
{
    printf("%s Starting...\n\n", argv[0]);
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    int n = 16;
    hiprandState_t *states;
    hipMalloc((void **) &states, n * sizeof(hiprandState_t));
    init_random<<<n,1>>>(time(0), states);

    double *host_nums = new double(n);
    double *device_nums;
    hipMalloc((void **) &device_nums, n * sizeof(double));

    randoms<<<n,1>>>(states, device_nums);

    hipMemcpy(host_nums, device_nums, n * sizeof(double), hipMemcpyDeviceToHost);

/*
    for (int i=0; i<n; i++)  {
        std::cout << host_nums[i] << "\n";
    }
*/
    free(host_nums);
    hipFree(states);
    hipFree(device_nums);

/*
    int n = 1024;
    double size = n * sizeof(double);
    double *gpu_x;
    hipMalloc((void **) &gpu_x, size);

    random<<<1,n>>>(gpu_x);

    double *x;
    x = (double *)malloc(size);
    hipMemcpy(x, gpu_x, size, hipMemcpyDeviceToHost);

    double total = 0.;
    for (int i=0; i<n; i++) {
        std::cout << x[i] << "\n";
        total += x[i];
    }

    std::cout << "\n" << total / double(n) << "\n";

    hipFree(gpu_x);
*/
}


// Program main
int main(int argc, char **argv) {
    pi(argc, argv);
}

