#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cstdlib>
#include <time.h>

// includes CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// declaration, forward
void pi(int argc, char **argv);

/******************************** DEVICE CODE ********************************/

__global__ void init_random(unsigned int seed, hiprandState_t *states)  {
    hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

__global__ void randoms(hiprandState_t *states, double *numbers)  {
    numbers[blockIdx.x] = hiprand_uniform_double(&states[blockIdx.x]);
}

/********************************* HOST CODE *********************************/

void pi(int argc, char **argv)
{
    printf("%s Starting...\n\n", argv[0]);
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    int n = 1024;
    hiprandState_t *states;
    hipMalloc((void **) &states, n * sizeof(hiprandState_t));
    init_random<<<n,1>>>(time(0), states);

    double *host_nums;
    host_nums = new double [n];
    double *device_nums;
    int size = n * sizeof(double);
    hipMalloc((void **) &device_nums, size);

    randoms<<<n,1>>>(states, device_nums);

    hipMemcpy(host_nums, device_nums, n * sizeof(double), hipMemcpyDeviceToHost);

    double total = 0;
    for (int i=0; i<n; i++)  {
        std::cout << i+1 << "    " << host_nums[i] << "\n";
        total += host_nums[i];
    }
    std::cout << "\naverage = " << total / double(n) << "\n";

    delete[] host_nums;
    hipFree(states);
    hipFree(device_nums);
}

// Program main
int main(int argc, char **argv) {
    pi(argc, argv);
}

/************************* stuff ***************************/
/*
__global__ void threadBlockAdd(int *a, int *b, int *c, int n)  {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index];
}
*/

/*
 * __global__ void random(double *result)  {

    hiprandState_t state;  // need a local state for each thread, see cs.umw.edu

    hiprand_init(1234ULL, threadIdx.x, 0, &state);
// arg1 -> the seed controls the sequence of random values that are produced
// arg2 -> sequence number is only important with multiple cores
// arg3 -> offset, how much extra we advance in the sequence for each call, can be 0
// hiprand works like rand - except that it takes a state as a parameter

    double tmp = hiprand_uniform_double(&state);
    result[threadIdx.x] = tmp;
}
*/

// some hints here
// http://stackoverflow.com/questions/11832202/cuda-random-number-generating
//
// this is pretty good...look at multi core implementation at bottom
// http://cs.umw.edu/~finlayson/class/fall16/cpsc425/notes/cuda-random.html
//
// good article
// http://stackoverflow.com/questions/26650391/generate-random-number-within-a-function-with-hiprand-without-preallocation



